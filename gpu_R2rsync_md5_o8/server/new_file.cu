#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <unistd.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sys/stat.h>
#include <getopt.h>
#include <stdint.h>
#include <vector>
#include <sys/types.h>
#include <math.h>
#include "hash.h"
#include "checksum.h"


void recalcu(int chunk_size, int chunk_num, int *stat, int jump_pos, int file_len, int total_threads, char *h_file, int *match_offset, int *match_chunkid, Node *ht, int j);
Node lookup_ht(Node *ht, int32 rc, int *chunk_id, uint &first_hashindex, uint &second_hashindex, uint &jump_time);
__device__ uint32 d_get_checksum1(char *buf1, int32 len, uint32 *d_s1, uint32 *d_s2);
__device__ void d_get_checksum2(const uint8_t *in, const size_t inlen, uint8_t *out);

__device__ uint d_hash(uint32 rc);
__device__ uint d_hash2(uint32 rc);
__device__ Node d_lookup_ht(Node *ht, int32 rc, int bucket_num, int c0, int c1, const int4* __restrict__ coef, int &pos, int thread_id);
__device__ bool d_char_compare(char *c1, char *c2);

__constant__ uint8_t k[16]={0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15};  

inline __device__ uint d_hash(uint32 rc){
    uint p =  1867;
    return (((rc>>16)& 0xffff ^ ((rc&0xffff) * p)) & 0xffff)%HASHSIZE;
}
inline __device__ uint d_hash2(uint32 x){
  x = ((x >> 16) ^ x) * 0x45d9f3b;
  x = ((x >> 16) ^ x) * 0x45d9f3b;
  x = (x >> 16) ^ x;
  return 1+x%(HASHSIZE-1);
}
inline __device__ bool d_char_compare(uint8_t *c1, uint8_t *c2){
  if(c1[0]!=c2[0] || c1[1]!=c2[1] || c1[2]!=c2[2] || c1[3]!=c2[3]) return false;
  else if(c1[4]!=c2[4] || c1[5]!=c2[5] || c1[6]!=c2[6] || c1[7]!=c2[7]) return false;
  else return true;
}
inline bool char_compare(uint8_t *c1, uint8_t *c2){
  if(c1[0]!=c2[0] || c1[1]!=c2[1] || c1[2]!=c2[2] || c1[3]!=c2[3]) return false;
  else if(c1[4]!=c2[4] || c1[5]!=c2[5] || c1[6]!=c2[6] || c1[7]!=c2[7]) return false;
  else return true;
}
__global__ void kernel_test(Node *d_ht, int *d_pos_array, int size){
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if(thread_id==0){
    for(int i=0;i<size;++i){
      int pos = d_pos_array[i];
      Node np = d_ht[pos];
      //printf("pos %d, np rc %d\n", pos, np.checksum);
    }
  }

}

__global__ void multiwarp_match(Node *ht, char *file, size_t file_len, int total_threads, int chunk_size, int chunk_num, 
          int *match_offset, int *match_chunkid, int *stat, int bucket_num, int c0, int c1, const int4* __restrict__ coef)
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int fileBeginPos = chunk_num*chunk_size*thread_id;
  int chunkBeginPos = chunk_num*thread_id;
  //printf("thread %d start from %d\n", thread_id, fileBeginPos);
  if(fileBeginPos < file_len){
    int recalcu = 1;
    uint32 rc;
    int match_num = 0;
    int i = 0;
    uint32 s1 = 0, s2 = 0;
    //the char in the head of a chunk, it can be used to store as the unmatch value and use to recalcu
    char chunk_head_value;
    for(; i < chunk_size*chunk_num;){
      //剩下的内容已经不够一个chunk_size
      if(fileBeginPos+i>file_len-chunk_size){
        break;
      }
      if(recalcu == 1) rc = d_get_checksum1(&file[fileBeginPos + i], chunk_size, &s1, &s2);
      else if(recalcu == 0){
        s1 -= chunk_head_value + CHAR_OFFSET; 
        s2 -= chunk_size * (chunk_head_value + CHAR_OFFSET);
        s1 += file[fileBeginPos+i+chunk_size-1] + CHAR_OFFSET;
        s2 += s1;
        rc = (s1 & 0xffff) + (s2 << 16);
      }
      
      chunk_head_value = file[fileBeginPos+i];
      int pos = 0;
      //printf("before hash look up, in thread %d, pos %d, the first chunk rc %d, ht pos is %d\n", thread_id, fileBeginPos + i, (int)rc, pos);
      Node np = d_lookup_ht(ht, rc, bucket_num, c0, c1, coef, pos, thread_id);
      //printf("after hash look up, in thread %d, pos %d, the first chunk rc %d, ht pos is %d\n", thread_id, fileBeginPos + i, (int)rc, pos);
      
      //recalcu = 0;
      //i += 1;
      if(np.checksum == rc){
        uint8_t sum2[8];
        d_get_checksum2((uint8_t*)&file[fileBeginPos+i], (size_t)chunk_size, (uint8_t*)sum2);
        //printf("First, thread %d have match rc\n", thread_id); 
        if(d_char_compare(sum2,np.md5)){  
          //printf("Ha, thread %d have match md5 at file pos %d\n", thread_id, fileBeginPos + i);          
          match_chunkid[chunkBeginPos + match_num] = np.chunk_id;
          match_offset[chunkBeginPos + match_num] = fileBeginPos + i;         
          match_num ++;
          recalcu = 1;
          i += chunk_size;
        }
        else{
          //printf("Ai, thread %d don't have match md5 at file pos %d\n", thread_id, fileBeginPos + i);       
          recalcu = 0;
          i += 1;
        }
      }
      else{
        /*if(thread_id==1){
          printf("Emmm, thread %d don't match at file pos %d, node rc %d, calcu rc %d\n", thread_id, fileBeginPos + i, np.checksum, rc); 
        }*/
        //printf("Emmm, thread %d don't match at file pos %d, node rc %d, calcu rc %d\n", thread_id, fileBeginPos + i, np.checksum, rc); 
        recalcu = 0;
        i += 1;
      }
    }
    stat[thread_id] = match_num;
  }
}

__global__ void gpu_recalcu(Node *ht, char *file, int chunk_size, int chunk_num, int *match_offset, int *match_chunkid,
             int *stat, int region_size, int bucket_num, int c0, int c1, const int4* __restrict__ coef)
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int start_t = thread_id * region_size;
  //printf("thread %d start recalcu on %d thread, region size %d\n", thread_id, start_t, region_size);
  for(int i=start_t; i<start_t+region_size-1; ++i){
    //printf("thread %d recalcu on its %d thread\n", thread_id, i-start_t);
    int t_match_num = stat[i];
    int j = i+1; 
    int jump_pos = match_offset[chunk_num*i+t_match_num-1]+chunk_size; 
    if(t_match_num > 0 && stat[j] > 0 && jump_pos > match_offset[chunk_num*j]){
      //if(i<10) printf("in gpu recalcu thread %d need recalcu, its match num %d, jump pos %d\n", i, t_match_num, jump_pos);         
      int match_index = 0;
      int recalcu = 1;
      int chunk_id;
      int j_match_num = stat[j];
      int j_match_begin = chunk_num*j;
      char chunk_head_value;
      uint32 s1, s2, rc;
      while(1){
        if(recalcu == 1) rc = d_get_checksum1(&file[jump_pos], chunk_size, &s1, &s2);   
        else if(recalcu == 0){
          s1 -= chunk_head_value + CHAR_OFFSET; 
          s2 -= chunk_size * (chunk_head_value + CHAR_OFFSET);
          s1 += file[jump_pos+chunk_size-1] + CHAR_OFFSET;
          s2 += s1;
          rc = (s1 & 0xffff) + (s2 << 16);
        }    
        while(jump_pos > match_offset[j_match_begin+match_index]){            
          if(match_index < j_match_num){
            match_chunkid[j_match_begin+match_index] = -1;
            stat[j]--;
            match_index++;
          } 
          else break;
        }
        if(jump_pos == match_offset[j_match_begin+match_index]) break;
        int pos;
        Node np = d_lookup_ht(ht, rc, bucket_num, c0, c1, coef, pos, thread_id);
        if(np.checksum == rc){
          uint8_t sum2[8];
          d_get_checksum2((uint8_t*)&file[jump_pos], (size_t)chunk_size, (uint8_t*)sum2);
          if(d_char_compare(sum2,np.md5)){  
            for(int k=j_match_begin;k<j_match_begin+chunk_num;++k){         
              if(match_chunkid[k]==-1 || jump_pos+chunk_size > match_offset[k]){
                match_offset[k] = jump_pos;
                match_chunkid[k] = chunk_id;
                stat[j]++;
                break;
              }
            }
            recalcu = 1;
            jump_pos += chunk_size;
          }
          else{
            chunk_head_value = file[jump_pos];
            recalcu = 0;
            jump_pos += 1;
          }
        }
        else{
          chunk_head_value = file[jump_pos];
          recalcu = 0;
          jump_pos += 1;
        } 
        if(match_index >= j_match_num) break;
      }
    }  
  }        
}




void recalcu(int chunk_size, int chunk_num, int *stat, int jump_pos, int file_len, int total_threads,
            char *h_file, int *match_offset, int *match_chunkid, Node *ht, int j, int recalcu_region_size){
  int match_index = 0;
  int unmatch_index = 0; // 
  int recalcu = 1;
  int chunk_id;
  int length = chunk_size;
  int j_match_num = 0;
  for(int i=0;i<recalcu_region_size;++i){
    j_match_num += stat[j+i];
  }
  int j_match_begin = chunk_num*j;
  char chunk_head_value;
  uint32 s1, s2, rc;
  while(1){
    if(recalcu == 1) rc = get_checksum1(&h_file[jump_pos], length, (int*)&s1, (int*)&s2);   
    else if(recalcu == 0){
      s1 -= chunk_head_value + CHAR_OFFSET; 
      s2 -= chunk_size * (chunk_head_value + CHAR_OFFSET);
      s1 += h_file[jump_pos+length-1] + CHAR_OFFSET;
      s2 += s1;
      rc = (s1 & 0xffff) + (s2 << 16);
    }
    while(jump_pos > match_offset[j_match_begin+match_index+unmatch_index]){
      if(match_chunkid[j_match_begin+match_index+unmatch_index] == -1){
        unmatch_index += 1;
      }
      else if(match_index < j_match_num){
        match_chunkid[j_match_begin+match_index+unmatch_index] = -1;
        //stat[j]--;
        match_index++;
      } 
      else break;
    }
    if(jump_pos == match_offset[j_match_begin+match_index+unmatch_index] && match_chunkid[j_match_begin+match_index+unmatch_index] != -1) break;
    
    uint first_index, second_index, jump_time;
    Node np = lookup_ht(ht, rc, &chunk_id, first_index, second_index, jump_time);
    if(np.chunk_id == -1){
      recalcu = 0;
      jump_pos += 1;
    }
    else{
      uint8_t sum2[8];
      get_checksum2((uint8_t*)&h_file[jump_pos], (size_t)length, (uint8_t*)sum2); 
      uint index3;
      for(int j = jump_time;;++j){   
        index3 = (first_index+j*second_index)%HASHSIZE;
        np = ht[index3];
        if(np.chunk_id == -1){
          recalcu = 0;
          chunk_head_value = h_file[jump_pos];
          jump_pos += 1;
          break;
        }
        if(char_compare(sum2,np.md5)){
          for(int k=j_match_begin;k<j_match_begin+chunk_num*recalcu_region_size;++k){   
            //已经被置为-1或者目前还没有但马上会被置为-1的       
            if(match_chunkid[k]==-1 || jump_pos+chunk_size > match_offset[k]){
              match_offset[k] = jump_pos;
              match_chunkid[k] = chunk_id;
              //stat[j]++;
              break;
            }
          }
          recalcu = 1;
          jump_pos += chunk_size;
          //printf("we have match in thread %d in gpu\n",thread_id);
          break;
        }      
      }  
    }     
    //还一种可能就是整个chunk_size*chunk_num都没有匹配
    if(match_index >= j_match_num) break;
    //printf("match_index is %d, j_match_num is %d\n",match_index, j_match_num);
  }
}

Node lookup_ht(Node *ht, int32 rc, int *chunk_id, uint &first_index, uint &second_index, uint &jump_time){ 
  uint index = hash(rc);
  uint index2 = hash2(rc);
  first_index = index;
  second_index = index2;
  uint index3;
  //这里是不一定能找到
  for(int i=0;;++i){
    index3 = (index+i*index2)%HASHSIZE;
    if(ht[index3].chunk_id == -1 || rc == ht[index3].checksum){
      jump_time = i;
      return ht[index3];
    }
  }
}


__device__ void d_get_checksum2(const uint8_t *in, const size_t inlen, uint8_t *out){
    uint64_t v0 = 0x736f6d6570736575ULL;
    uint64_t v1 = 0x646f72616e646f6dULL;
    uint64_t v2 = 0x6c7967656e657261ULL;
    uint64_t v3 = 0x7465646279746573ULL;
    //uint64_t k0 = 50462976;
    //uint64_t k1 = 185207048;
    uint64_t k0 = U8TO64_LE(k);
    uint64_t k1 = U8TO64_LE(k + 8);
    uint64_t m;
    const uint8_t *end = in + inlen - (inlen % sizeof(uint64_t));
    const int left = inlen & 7;
    uint64_t b = ((uint64_t)inlen) << 56;
    v3 ^= k1;
    v2 ^= k0;
    v1 ^= k1;
    v0 ^= k0;

    for (; in != end; in += 8) {
        m = U8TO64_LE(in);
        v3 ^= m;
        SIPROUND;
        SIPROUND;
        v0 ^= m;
    }

    switch (left) {
    case 7:
        b |= ((uint64_t)in[6]) << 48;
    case 6:
        b |= ((uint64_t)in[5]) << 40;
    case 5:
        b |= ((uint64_t)in[4]) << 32;
    case 4:
        b |= ((uint64_t)in[3]) << 24;
    case 3:
        b |= ((uint64_t)in[2]) << 16;
    case 2:
        b |= ((uint64_t)in[1]) << 8;
    case 1:
        b |= ((uint64_t)in[0]);
        break;
    case 0:
        break;
    }
    v3 ^= b;
    SIPROUND;
    SIPROUND;
    v0 ^= b;
    v2 ^= 0xff;
    SIPROUND;
    SIPROUND;
    SIPROUND;
    SIPROUND;
    b = v0 ^ v1 ^ v2 ^ v3;
    U64TO8_LE(out, b);
}



__device__ uint32 d_get_checksum1(char *buf1, int32 len, uint32 *d_s1, uint32 *d_s2)
{
    int32 i;
    uint32 s1, s2;
    char *buf = (char *)buf1;
    s1 = s2 = 0;
    for (i = 0; i < (len-4); i+=4) {
        s2 += 4*(s1 + buf[i]) + 3*buf[i+1] + 2*buf[i+2] + buf[i+3] +
          10*CHAR_OFFSET;
        s1 += (buf[i+0] + buf[i+1] + buf[i+2] + buf[i+3] + 4*CHAR_OFFSET);
    }
    for (; i < len; i++) {
        s1 += (buf[i]+CHAR_OFFSET); s2 += s1;
    }
    *d_s1 = s1;
    *d_s2 = s2;
    return (s1 & 0xffff) + (s2 << 16);
}


__device__ Node d_lookup_ht(Node *ht, int32 rc, int bucket_num, int c0, int c1, const int4* __restrict__ coef, int &pos, int thread_id){ 
  unsigned long long int step00 = c0 + c1*abs(rc);
  int bucket_id = ((step00)%1900813+1900813)%bucket_num;
  int4 i4 = coef[bucket_id];
  unsigned long long int step01 = (i4.y+i4.z*(abs(rc)));
  int node_pos = (step01%1900813+1900813)%i4.x + i4.w;
  //printf("GPU bucket id %d, rc %d, step0_pos %lld, step1_pos %d, node pos %d\n", bucket_id, rc, step0_pos, step1_pos, node_pos);
  if(thread_id == 0){
    printf("GPU first roudn rc %d, c0 %d, c1 %d, bucket_num %d, step00 %lld, bucket id %d\n", rc, c0, c1, bucket_num, step00, bucket_id);
  }
  //printf("GPU bucket id %d, rc %d, step01_pos %lld, node pos %d\n", bucket_id, rc, step01, node_pos);
  pos = node_pos;
  //printf("GPU node pos %d checksum %d\n",node_pos, ht[node_pos].checksum);
  return ht[node_pos];
}
