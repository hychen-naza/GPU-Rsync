#include <set>
#include <map>
#include <iostream>
#include "hash.h"

uint hash(uint32 rc){
    uint p =  1867;
    return (((rc>>16)& 0xffff ^ ((rc&0xffff) * p)) & 0xffff)%HASHSIZE;
}

hipStream_t * GPUWarmUp(int n_stream)
{
  Node *p;
  hipMallocManaged(&p,sizeof(Node));
  int dev = 0;
  hipSetDevice(dev);
  hipStream_t *stream =(hipStream_t*)malloc(n_stream*sizeof(hipStream_t));
  for(int i=0;i<n_stream;i++)
  {
    hipStreamCreate(&stream[i]);
  }
  return stream;
}

bool IsSameChunk(Node *p, uint id, uint32 checksum, uint8_t md5[8], std::vector<std::vector<int> > &matchIdVec){
  if(p->chunk_id == -1) return false;
  if(p->checksum != checksum) return false;
  if(memcmp(p->md5, md5, 8) != 0) return false;
  matchIdVec[p->chunk_id].push_back(id);
  std::cout << "we find a same chunk , it is rare\n";
  return true;
}

int insert_hashtable(Node *ht, uint id, uint32 checksum, uint8_t md5[8], std::vector<std::vector<int> > &matchIdVec)
{
  uint index = hash(checksum);
  uint i = 0;
  if(ht[index].chunk_id == -1){
    ht[index].chunk_id = id;
    ht[index].checksum = checksum;
    memcpy(ht[index].md5, md5, 8);
    ht[index].next = NULL;
    matchIdVec[id].push_back(id);
    return 1;
  }
  else{
    Node *p = &ht[index];
    for(;p != NULL; p=(p->next)){
      if(IsSameChunk(p, id, checksum, md5, matchIdVec)) return 1;
      if(p->next == NULL){
        p->next = (Node *)malloc(sizeof(Node));
        hipMallocManaged(&(p->next),sizeof(Node));
        p->next->chunk_id = id;
        p->next->checksum = checksum;
        memcpy(p->next->md5, md5, 8);
        p->next->next = NULL;
        matchIdVec[id].push_back(id);
        return 1;   
      }
    }    
  }
  return 0;  
}











