#include <set>
#include <map>
#include <iostream>
#include "hash.h"

uint hash(uint32 rc){
    uint p =  1867;
    return (((rc>>16)& 0xffff ^ ((rc&0xffff) * p)) & 0xffff)%HASHSIZE;
}

hipStream_t * GPUWarmUp(int n_stream)
{
  Node *p;
  hipMallocManaged(&p,sizeof(Node));
  int dev = 0;
  hipSetDevice(dev);
  hipStream_t *stream =(hipStream_t*)malloc(n_stream*sizeof(hipStream_t));
  for(int i=0;i<n_stream;i++)
  {
    hipStreamCreate(&stream[i]);
  }
  return stream;
}

bool IsSameChunk(Node *p, uint id, uint32 checksum, char md5[16], int *matchIdArray, std::map<int, std::set<int> > &matchIdMap){
  if(p->chunk_id == -1) return false;
  if(p->checksum != checksum) return false;
  for(int i=0;i<16;++i){
    if(p->md5[i] != md5[i]){
      return false;
    }
  }
  int original_id = p->chunk_id;
  matchIdArray[original_id] = -1;
  matchIdMap[original_id].insert(original_id);
  matchIdMap[original_id].insert(id);
  //std::cout << "we find a same chunk , it is rare\n";
  return true;
}

int insert_hashtable(Node *ht, uint id, uint32 checksum, char md5[16], int *matchIdArray, std::map<int, std::set<int> > &matchIdMap)
{
  uint index = hash(checksum);
  uint i = 0;
  if(ht[index].chunk_id == -1){
    ht[index].chunk_id = id;
    ht[index].checksum = checksum;
    for(i=0;i<16;++i){
      ht[index].md5[i] = md5[i];
    }
    ht[index].next = NULL;
    matchIdArray[id] = id;
    return 1;
  }
  else{
    Node *p = &ht[index];
    for(;p != NULL; p=(p->next)){
      if(IsSameChunk(p, id, checksum, md5, matchIdArray, matchIdMap)) return 1;
      if(p->next == NULL){
        p->next = (Node *)malloc(sizeof(Node));
        hipMallocManaged(&(p->next),sizeof(Node));
        p->next->chunk_id = id;
        p->next->checksum = checksum;
        for(i=0;i<16;++i){
          p->next->md5[i] = md5[i];
        }
        p->next->next = NULL;
        matchIdArray[id] = id;
        return 1;   
      }
    }    
  }
  return 0;  
}











